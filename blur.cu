#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define IMAGE_HEIGHT 521
#define IMAGE_WIDTH 428

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void blur(int *d_R, int *d_G, int *d_B, int *d_Rnew, int *d_Gnew, int *d_Bnew)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT) {
      return;
  }
  if (y != 0 && y != (IMAGE_HEIGHT-1) && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/4;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/4;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/4;
  }
  else if (y == 0 && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (y == (IMAGE_HEIGHT-1) && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (x == 0 && y != 0 && y != (IMAGE_HEIGHT-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1])/3;
  }
  else if (x == (IMAGE_WIDTH-1) && y != 0 && y != (IMAGE_HEIGHT-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (y==0 &&x==0){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y+1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y+1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y+1) + x])/2;
  }
  else if (y==0 &&x==(IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x-1]+d_R[(IMAGE_WIDTH*y+1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x-1]+d_G[(IMAGE_WIDTH*y+1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x-1]+d_B[(IMAGE_WIDTH*y+1) + x])/2;
  }
  else if (y==(IMAGE_HEIGHT-1) &&x==0){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y-1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y-1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y-1) + x])/2;
  }
  else if (y==(IMAGE_HEIGHT-1) &&x==(IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x-1]+d_R[(IMAGE_WIDTH*y-1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x-1]+d_G[(IMAGE_WIDTH*y-1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x-1]+d_B[(IMAGE_WIDTH*y-1) + x])/2;
  }

}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int Rnew[rowsize][colsize], Gnew[rowsize][colsize], Bnew[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	gettimeofday(&tim, NULL);

	fp = fopen("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);

	nblurs = 20;
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	for(k=0;k<nblurs;k++){
		int flat_R[IMAGE_WIDTH * IMAGE_HEIGHT];
		int flat_G[IMAGE_WIDTH * IMAGE_HEIGHT];
		int flat_B[IMAGE_WIDTH * IMAGE_HEIGHT];
		for (int row=0;row<IMAGE_HEIGHT;row++){
			for (int col=0;col<IMAGE_WIDTH;col++){
				flat_R[IMAGE_WIDTH*row+col] = R[row][col];
				flat_G[IMAGE_WIDTH*row+col] = G[row][col];
				flat_B[IMAGE_WIDTH*row+col] = B[row][col];
			}
		}
		int *d_R, *d_G, *d_B, *d_Rnew, *d_Gnew, *d_Bnew;
		int size = sizeof(int) * IMAGE_WIDTH * IMAGE_HEIGHT;
		hipMalloc((void **)&d_R, size);
		hipMemcpy(d_R, flat_R, size, hipMemcpyHostToDevice);
		hipMalloc((void **)&d_G, size);
		hipMemcpy(d_G, flat_G, size, hipMemcpyHostToDevice);
		hipMalloc((void **)&d_B, size);
    hipMemcpy(d_B, flat_B, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_Rnew, size);
		hipMalloc((void **)&d_Bnew, size);
    hipMalloc((void **)&d_Gnew, size);
		int numBlocksY = ceil(IMAGE_HEIGHT/16.0);
		int numBlocksX = ceil(IMAGE_WIDTH/16.0);
		dim3 dimBlock(numBlocksX,numBlocksY);
		dim3 dimGrid(16, 16);


    // Punch it Chewie
		blur<<<dimGrid, dimBlock>>>(d_R, d_G, d_B, d_Rnew, d_Gnew, d_Bnew);


		int *h_R, *h_G, *h_B;
		h_R = (int *)malloc(size);
    h_G = (int *)malloc(size);
		h_B = (int *)malloc(size);
		hipMemcpy(h_R, d_Rnew, size, hipMemcpyDeviceToHost);
		hipMemcpy(h_G, d_Gnew, size, hipMemcpyDeviceToHost);
		hipMemcpy(h_B, d_Bnew, size, hipMemcpyDeviceToHost);
		hipFree(d_R);
		hipFree(d_G);
		hipFree(d_B);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		for (int row=0;row<IMAGE_HEIGHT;row++){
			for (int col=0;col<IMAGE_WIDTH;col++){
				R[row][col] = h_R[IMAGE_WIDTH*row+col];
				G[row][col] = h_G[IMAGE_WIDTH*row+col];
				B[row][col] = h_B[IMAGE_WIDTH*row+col];
			}
		}

		// for(row=0;row<rowsize;row++){
		// 	for (col=0;col<colsize;col++){
		// 	    R[row][col] = Rnew[row][col];
		// 	    G[row][col] = Gnew[row][col];
		// 	    B[row][col] = Bnew[row][col];
		// 	}
		// }
	}

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
