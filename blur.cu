#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define IMAGE_WIDTH 521
#define IMAGE_HEIGHT 428

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void blur(int *d_R, int *d_O)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT) {
      return;
  }
  int myval = d_R[(IMAGE_WIDTH*x) + y];
  d_O[(IMAGE_WIDTH*x) + y] = 5;
//   printf("%d\n", sizeof(d_R)/sizeof(int));
}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int Rnew[rowsize][colsize], Gnew[rowsize][colsize], Bnew[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	gettimeofday(&tim, NULL);

	fp = fopen("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);

	nblurs = 1;
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	for(k=0;k<nblurs;k++){
		int flat_R[rowsize * colsize];
		for (int row=0;row<colsize;row++){
			for (int col=0;col<rowsize;col++){
				flat_R[rowsize*row+col] = R[col][row];
			}
		}
		int *d_R;
		int size = sizeof(int) * colsize * rowsize;
		// printf("%d\n", size);
		hipMalloc((void **)&d_R, size);
		hipMemcpy(d_R, flat_R, size, hipMemcpyHostToDevice);
		int *d_O;
		hipMalloc((void **)&d_O, size);
		int numBlocksY = ceil(colsize/16.0);
		int numBlocksX = ceil(rowsize/16.0);
		dim3 dimBlock(numBlocksX,numBlocksY);
		dim3 dimGrid(16, 16);

		blur<<<dimGrid, dimBlock>>>(d_R, d_O);
		int *h_R;
		h_R = (int *)malloc(size);
		hipMemcpy(h_R, d_O, size, hipMemcpyDeviceToHost);
		hipFree(d_R);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		for (int row=0;row<colsize;row++){
			for (int col=0;col<rowsize;col++){
				printf("%d\n", h_R[rowsize*row+col]);
				R[col][row] = h_R[rowsize*row+col];
			}
		}

		// for(row=0;row<rowsize;row++){
		// 	for (col=0;col<colsize;col++){
		// 	    R[row][col] = Rnew[row][col];
		// 	    G[row][col] = Gnew[row][col];
		// 	    B[row][col] = Bnew[row][col];
		// 	}
		// }
	}

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
