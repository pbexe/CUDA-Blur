#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define IMAGE_WIDTH 428
#define IMAGE_HEIGHT 521

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void blur(int *d_R,int *d_G, int *d_B)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT) {
      return;
  }
  int myval = d_R[(IMAGE_WIDTH*x) + y];
  d_R[(IMAGE_WIDTH*x) + y] = d_R[(IMAGE_WIDTH*x) + y] / 2;
  d_G[(IMAGE_WIDTH*x) + y] = d_G[(IMAGE_WIDTH*x) + y] / 2;
  d_B[(IMAGE_WIDTH*x) + y] = d_B[(IMAGE_WIDTH*x) + y] / 2;
}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int Rnew[rowsize][colsize], Gnew[rowsize][colsize], Bnew[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	gettimeofday(&tim, NULL);

	fp = fopen("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);

	nblurs = 1;
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	for(k=0;k<nblurs;k++){
		int flat_R[rowsize * colsize];
		int flat_G[rowsize * colsize];
		int flat_B[rowsize * colsize];
		for (int row=0;row<rowsize;row++){
			for (int col=0;col<colsize;col++){
				flat_R[colsize*row+col] = R[row][col];
				flat_G[colsize*row+col] = G[row][col];
				flat_B[colsize*row+col] = B[row][col];
			}
		}
		int *d_R, *d_G, *d_B;
		int size = sizeof(int) * colsize * rowsize;
		// printf("%d\n", size);
		hipMalloc((void **)&d_R, size);
		hipMemcpy(d_R, flat_R, size, hipMemcpyHostToDevice);
		hipMalloc((void **)&d_G, size);
		hipMemcpy(d_G, flat_G, size, hipMemcpyHostToDevice);
		hipMalloc((void **)&d_B, size);
		hipMemcpy(d_B, flat_B, size, hipMemcpyHostToDevice);
		int numBlocksY = ceil(colsize/16.0);
		int numBlocksX = ceil(rowsize/16.0);
		dim3 dimBlock(numBlocksY,numBlocksX);
		dim3 dimGrid(16, 16);


    // Punch it chewie
		blur<<<dimGrid, dimBlock>>>(d_R, d_G, d_B);


		int *h_R, *h_G, *h_B;
		h_R = (int *)malloc(size);
		h_G = (int *)malloc(size);
		h_B = (int *)malloc(size);
		hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
		hipMemcpy(h_G, d_G, size, hipMemcpyDeviceToHost);
		hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		hipFree(d_R);
		hipFree(d_G);
		hipFree(d_B);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		for (int row=0;row<rowsize;row++){
			for (int col=0;col<colsize;col++){
				R[row][col] = h_R[colsize*row+col];
				G[row][col] = h_G[colsize*row+col];
				B[row][col] = h_B[colsize*row+col];
			}
		}

		// for(row=0;row<rowsize;row++){
		// 	for (col=0;col<colsize;col++){
		// 	    R[row][col] = Rnew[row][col];
		// 	    G[row][col] = Gnew[row][col];
		// 	    B[row][col] = Bnew[row][col];
		// 	}
		// }
	}

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
