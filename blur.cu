#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define IMAGE_HEIGHT 521
#define IMAGE_WIDTH 428

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void blur(int *d_R, int *d_G, int *d_B, int *d_Rnew, int *d_Gnew, int *d_Bnew)
{
  // Get the X and y coords of the pixel for this thread
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  // Stop the thread if it is not part of the image
	if (x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT) {
      return;
  }

  // Apply the box blur
  if (y != 0 && y != (IMAGE_HEIGHT-1) && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/4;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/4;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/4;
  }
  else if (y == 0 && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (y == (IMAGE_HEIGHT-1) && x != 0 && x != (IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (x == 0 && y != 0 && y != (IMAGE_HEIGHT-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x+1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x+1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x+1])/3;
  }
  else if (x == (IMAGE_WIDTH-1) && y != 0 && y != (IMAGE_HEIGHT-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y+1) + x]+d_R[(IMAGE_WIDTH*y-1) + x]+d_R[(IMAGE_WIDTH*y) + x-1])/3;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y+1) + x]+d_G[(IMAGE_WIDTH*y-1) + x]+d_G[(IMAGE_WIDTH*y) + x-1])/3;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y+1) + x]+d_B[(IMAGE_WIDTH*y-1) + x]+d_B[(IMAGE_WIDTH*y) + x-1])/3;
  }
  else if (y==0 &&x==0){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y+1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y+1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y+1) + x])/2;
  }
  else if (y==0 &&x==(IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x-1]+d_R[(IMAGE_WIDTH*y+1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x-1]+d_G[(IMAGE_WIDTH*y+1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x-1]+d_B[(IMAGE_WIDTH*y+1) + x])/2;
  }
  else if (y==(IMAGE_HEIGHT-1) &&x==0){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x+1]+d_R[(IMAGE_WIDTH*y-1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x+1]+d_G[(IMAGE_WIDTH*y-1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x+1]+d_B[(IMAGE_WIDTH*y-1) + x])/2;
  }
  else if (y==(IMAGE_HEIGHT-1) &&x==(IMAGE_WIDTH-1)){
    d_Rnew[(IMAGE_WIDTH*y) + x] = (d_R[(IMAGE_WIDTH*y) + x-1]+d_R[(IMAGE_WIDTH*y-1) + x])/2;
    d_Gnew[(IMAGE_WIDTH*y) + x] = (d_G[(IMAGE_WIDTH*y) + x-1]+d_G[(IMAGE_WIDTH*y-1) + x])/2;
    d_Bnew[(IMAGE_WIDTH*y) + x] = (d_B[(IMAGE_WIDTH*y) + x-1]+d_B[(IMAGE_WIDTH*y-1) + x])/2;
  }

}

int main (int argc, const char * argv[]) {
  struct timeval tim;
	gettimeofday(&tim, NULL);
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;

	fp = fopen("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
  // Number of blur iterations
	nblurs = atoi(argv[1]);
  // Start the timer
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
  // The size of the 1D arrays for the GPU
  int size = sizeof(int) * IMAGE_WIDTH * IMAGE_HEIGHT;
  // Initialise the arrays to hold the flatened image
  int *flat_R, *flat_G, *flat_B;
  flat_R = (int *)malloc(size);
  flat_G = (int *)malloc(size);
  flat_B = (int *)malloc(size);
  // Create pointers to GPU array locations
  int *d_R, *d_G, *d_B, *d_Rnew, *d_Gnew, *d_Bnew;
  // Define how many threads per block
  int numBlocksY = ceil(IMAGE_HEIGHT/16.0);
  int numBlocksX = ceil(IMAGE_WIDTH/16.0);
  dim3 dimBlock(numBlocksX,numBlocksY);
  // Define how many blocks per grid
  dim3 dimGrid(16, 16);
  // Allocate GPU mem for the 1D arrays
  hipMalloc((void **)&d_R, size);
  hipMalloc((void **)&d_G, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_Rnew, size);
  hipMalloc((void **)&d_Bnew, size);
  hipMalloc((void **)&d_Gnew, size);
  // Pointers to handle the output
  int *h_R, *h_G, *h_B;
  h_R = (int *)malloc(size);
  h_G = (int *)malloc(size);
  h_B = (int *)malloc(size);
  // Start the blur loop
  for(k=0;k<nblurs;k++){
    // Flatten the 2D arrays to make them easier to handle with CUDA
  	for (int row=0;row<IMAGE_HEIGHT;row++){
  		for (int col=0;col<IMAGE_WIDTH;col++){
  			flat_R[IMAGE_WIDTH*row+col] = R[row][col];
  			flat_G[IMAGE_WIDTH*row+col] = G[row][col];
  			flat_B[IMAGE_WIDTH*row+col] = B[row][col];
  		}
  	}
    // Copy these arrays to the GPU
  	hipMemcpy(d_R, flat_R, size, hipMemcpyHostToDevice);
  	hipMemcpy(d_G, flat_G, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, flat_B, size, hipMemcpyHostToDevice);


    // Punch it Chewie
  	blur<<<dimGrid, dimBlock>>>(d_R, d_G, d_B, d_Rnew, d_Gnew, d_Bnew);

    // Copy the modified values out of the GPU
  	hipMemcpy(h_R, d_Rnew, size, hipMemcpyDeviceToHost);
  	hipMemcpy(h_G, d_Gnew, size, hipMemcpyDeviceToHost);
  	hipMemcpy(h_B, d_Bnew, size, hipMemcpyDeviceToHost);
    // Check for errors
  	gpuErrchk( hipPeekAtLastError() );
  	gpuErrchk( hipDeviceSynchronize() );
    // Convert the 1D arrays back into 2D
  	for (int row=0;row<IMAGE_HEIGHT;row++){
  		for (int col=0;col<IMAGE_WIDTH;col++){
  			R[row][col] = h_R[IMAGE_WIDTH*row+col];
  			G[row][col] = h_G[IMAGE_WIDTH*row+col];
  			B[row][col] = h_B[IMAGE_WIDTH*row+col];
  		}
  	}
  }
  // Free up the allocated memory
  hipFree(d_R);
  hipFree(d_G);
  hipFree(d_B);
  hipFree(d_Rnew);
  hipFree(d_Gnew);
  hipFree(d_Bnew);
  free(h_R);
  free(h_G);
  free(h_B);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
  gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);
  return 0;
}
